#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <iomanip>

// Color codes for formatting
#define RESET "\033[0m"
#define BOLD "\033[1m"
#define RED "\033[31m"
#define GREEN "\033[32m"
#define YELLOW "\033[33m"
#define BLUE "\033[34m"
#define MAGENTA "\033[35m"
#define CYAN "\033[36m"
#define WHITE "\033[37m"

// Bold colors
#define BOLDRED "\033[1;31m"
#define BOLDGREEN "\033[1;32m"
#define BOLDYELLOW "\033[1;33m"
#define BOLDBLUE "\033[1;34m"
#define BOLDMAGENTA "\033[1;35m"
#define BOLDCYAN "\033[1;36m"
#define BOLDWHITE "\033[1;37m"

// Format functions for memory size, frequency, and dimensions of threads and blocks for human-readable output
std::string formatMemorySize(size_t bytes)
{
       const char *units[] = {"B", "KB", "MB", "GB", "TB"};
       int unitIndex = 0;
       double size = static_cast<double>(bytes);

       while (size >= 1024 && unitIndex < 4)
       {
              size /= 1024;
              unitIndex++;
       }

       std::stringstream ss;
       ss << std::fixed << std::setprecision(2) << size << " " << units[unitIndex];
       return ss.str();
}

std::string formatFrequency(int freqInKHz)
{
       if (freqInKHz >= 1000000)
       {
              return std::to_string(freqInKHz / 1000000) + " GHz";
       }
       else if (freqInKHz >= 1000)
       {
              return std::to_string(freqInKHz / 1000) + " MHz";
       }
       return std::to_string(freqInKHz) + " kHz";
}

std::string formatDimension(int x, int y = 0, int z = 0)
{
       std::stringstream ss;
       if (z > 0)
       {
              ss << x << " x " << y << " x " << z;
       }
       else if (y > 0)
       {
              ss << x << " x " << y;
       }
       else
       {
              ss << x;
       }
       return ss.str();
}

int getCudaCoresPerSM(int major, int minor)
{
       switch (major)
       {
       case 2:
              return 32; // Fermi
       case 3:
              return 192; // Kepler
       case 5:
              return 128; // Maxwell
       case 6:
              return 64; // Pascal
       case 7:
              return 64; // Volta/Turing
       case 8:
              return 128; // Ampere
       case 9:
              return 128; // Hopper
       default:
              return 0;
       }
}
std::string getArchitectureName(int major, int minor)
{
       if (major == 2)
              return "Fermi";
       if (major == 3)
              return "Kepler";
       if (major == 5)
              return "Maxwell";
       if (major == 6)
              return "Pascal";
       if (major == 7)
       {
              if (minor == 0)
                     return "Volta";
              if (minor == 5)
                     return "Turing";
       }
       if (major == 8)
       {
              if (minor == 0)
                     return "Ampere";
              if (minor >= 6)
                     return "Ada Lovelace";
       }
       if (major == 9)
              return "Hopper";
       return "Unknown";
}
// Functions for printing headers, sub-headers, and key-value pairs
void printHeader(const char *header)
{
       printf("\n%s%s%s\n", BOLDCYAN, header, RESET);
}

void printSubHeader(const char *header)
{
       printf("%s%s%s\n", BOLDYELLOW, header, RESET);
}

void printValue(const char *label, const std::string &value)
{
       printf("%s%-45s%s%s%s\n", CYAN, label, RESET, value.c_str(), RESET);
}

void printValueBool(const char *label, bool value)
{
       printf("%s%-45s%s%s%s\n", CYAN, label, RESET, value ? "Yes" : "No", RESET);
}

// Format TFLOPS value as a string
std::string formatTFlops(float tflops)
{
       std::stringstream ss;
       ss << std::fixed << std::setprecision(2) << tflops << " TFLOPS";
       return ss.str();
}

// Format percentage value as a string
std::string formatPercentage(float value)
{
       std::stringstream ss;
       ss << std::fixed << std::setprecision(2) << value << "%";
       return ss.str();
}

int main(void)
{
       int deviceCount;
       hipGetDeviceCount(&deviceCount);

       if (deviceCount == 0)
       {
              printf("%sNo CUDA devices found.%s\n", BOLDRED, RESET);
              return EXIT_FAILURE;
       }

       for (int deviceIndex = 0; deviceIndex < deviceCount; ++deviceIndex)
       {
              hipDeviceProp_t deviceProp;
              hipGetDeviceProperties(&deviceProp, deviceIndex);

              // Device Header
              printf("\n%s%s========== CUDA Device #%d Information ==========%s\n",
                     BOLDMAGENTA, BOLD, deviceIndex, RESET);

              // Basic Device Information
              printHeader("Basic Device Information:");
              printValue("Device Name:", deviceProp.name);
              printValue("Compute Capability:", std::to_string(deviceProp.major) + "." + std::to_string(deviceProp.minor) + " (" + getArchitectureName(deviceProp.major, deviceProp.minor) + ")");
              printValue("MultiProcessor Count:",
                         std::to_string(deviceProp.multiProcessorCount) + " SMs");
              printValue("Maximum Threads Per MultiProcessor:",
                         std::to_string(deviceProp.maxThreadsPerMultiProcessor) + " threads");

              printValue("CUDA Cores per SM:", std::to_string(getCudaCoresPerSM(deviceProp.major, deviceProp.minor)));
              printValue("Total CUDA Cores:", std::to_string(getCudaCoresPerSM(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount));
              printValue("Device Compute Mode:", deviceProp.computeMode == hipComputeModeDefault            ? "Default"
                                                 : deviceProp.computeMode == hipComputeModeExclusive        ? "Exclusive"
                                                 : deviceProp.computeMode == hipComputeModeProhibited       ? "Prohibited"
                                                 : deviceProp.computeMode == hipComputeModeExclusiveProcess ? "Exclusive Process"
                                                                                                             : "Unknown");

              printValueBool("Device Integrated:", deviceProp.integrated);
              printValueBool("Device TCC Driver:", deviceProp.tccDriver);
              printValue("CUDA Driver Version:", std::to_string([]
                                                                { int driverVersion; hipDriverGetVersion(&driverVersion); return driverVersion; }()));
              printValue("Max Shared Memory Per Block:", formatMemorySize(deviceProp.sharedMemPerBlock));
              printValue("Max Grid Size:", formatDimension(deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]));
              printValue("Max Texture Gather Size:", std::to_string(deviceProp.maxTexture2DGather[0]) + " x " + std::to_string(deviceProp.maxTexture2DGather[1]));
              printValueBool("Unified Memory Support:", deviceProp.unifiedAddressing);

              // Memory Information
              printHeader("Memory Information:");
              printValue("Total Global Memory:", formatMemorySize(deviceProp.totalGlobalMem));
              printValue("Total Constant Memory:", formatMemorySize(deviceProp.totalConstMem));
              printValue("Shared Memory Per Block:", formatMemorySize(deviceProp.sharedMemPerBlock));
              printValue("Reserved Shared Memory Per Block:",
                         formatMemorySize(deviceProp.reservedSharedMemPerBlock));
              printValue("Memory Bus Width:", std::to_string(deviceProp.memoryBusWidth) + " bits");
              printValue("Memory Clock Rate:", formatFrequency(deviceProp.memoryClockRate));
              printValue("L2 Cache Size:", formatMemorySize(deviceProp.l2CacheSize));
              printValueBool("Global L1 Cache Supported:", deviceProp.globalL1CacheSupported);
              printValueBool("Local L1 Cache Supported:", deviceProp.localL1CacheSupported);
              printValueBool("Pageable Memory Access:", deviceProp.pageableMemoryAccess);
              printValueBool("Concurrent Managed Memory:", deviceProp.concurrentManagedAccess);

              // Advanced Memory Properties
              printHeader("Advanced Memory Properties:");
              printValue("Max Registers Per Block:", std::to_string(deviceProp.regsPerBlock));
#if CUDART_VERSION >= 12000
              printValueBool("Memory Pools Supported:", deviceProp.memoryPoolsSupported);
#endif
              printValue("Access Policy Max Window Size:",
                         formatMemorySize(deviceProp.accessPolicyMaxWindowSize));
              printValueBool("Host Register Supported:", deviceProp.hostRegisterSupported);
              printValueBool("Direct Managed Memory Access:",
                             deviceProp.directManagedMemAccessFromHost);

              // Thread and Block Information
              printHeader("Thread and Block Information:");
              printValue("Max Threads Per Block:",
                         std::to_string(deviceProp.maxThreadsPerBlock) + " threads");
              printValue("Max Thread Dimensions:",
                         formatDimension(deviceProp.maxThreadsDim[0],
                                         deviceProp.maxThreadsDim[1],
                                         deviceProp.maxThreadsDim[2]) +
                             " threads");
              printValue("Max Grid Dimensions:",
                         formatDimension(deviceProp.maxGridSize[0],
                                         deviceProp.maxGridSize[1],
                                         deviceProp.maxGridSize[2]) +
                             " blocks");
              printValue("Warp Size:", std::to_string(deviceProp.warpSize) + " threads");

              // Clock Information
              printHeader("Clock Information:");
              printValue("Clock Rate:", formatFrequency(deviceProp.clockRate));

              // Texture Information
              printHeader("Texture Memory Information:");
              printValue("Maximum 1D Texture Size:",
                         std::to_string(deviceProp.maxTexture1D) + " texels");
              printValue("Maximum 2D Texture Dimensions:",
                         formatDimension(deviceProp.maxTexture2D[0],
                                         deviceProp.maxTexture2D[1]) +
                             " texels");
              printValue("Maximum 3D Texture Dimensions:",
                         formatDimension(deviceProp.maxTexture3D[0],
                                         deviceProp.maxTexture3D[1],
                                         deviceProp.maxTexture3D[2]) +
                             " texels");
              printValue("Maximum 1D Layered Texture Size:",
                         formatDimension(deviceProp.maxTexture1DLayered[0],
                                         deviceProp.maxTexture1DLayered[1]) +
                             " texels");
              printValue("Maximum 2D Layered Texture Size:",
                         formatDimension(deviceProp.maxTexture2DLayered[0],
                                         deviceProp.maxTexture2DLayered[1],
                                         deviceProp.maxTexture2DLayered[2]) +
                             " texels");
              printValue("Texture Alignment:", formatMemorySize(deviceProp.textureAlignment));

              // Surface Information
              printHeader("Surface Memory Information:");
              printValue("Maximum 1D Surface Size:",
                         std::to_string(deviceProp.maxSurface1D) + " elements");
              printValue("Maximum 2D Surface Dimensions:",
                         formatDimension(deviceProp.maxSurface2D[0],
                                         deviceProp.maxSurface2D[1]) +
                             " elements");
              printValue("Maximum 3D Surface Dimensions:",
                         formatDimension(deviceProp.maxSurface3D[0],
                                         deviceProp.maxSurface3D[1],
                                         deviceProp.maxSurface3D[2]) +
                             " elements");
              printValue("Maximum 1D Layered Surface Size:",
                         formatDimension(deviceProp.maxSurface1DLayered[0],
                                         deviceProp.maxSurface1DLayered[1]) +
                             " elements");
              printValue("Maximum 2D Layered Surface Size:",
                         formatDimension(deviceProp.maxSurface2DLayered[0],
                                         deviceProp.maxSurface2DLayered[1],
                                         deviceProp.maxSurface2DLayered[2]) +
                             " elements");
              printValue("Surface Alignment:", formatMemorySize(deviceProp.surfaceAlignment));

              // Advanced Features
              printHeader("Advanced Features:");
              printValueBool("Concurrent Kernels:", deviceProp.concurrentKernels);
              printValueBool("Device Overlap Support:", deviceProp.deviceOverlap);
              printValue("Async Engine Count:", std::to_string(deviceProp.asyncEngineCount));
              printValueBool("Unified Addressing:", deviceProp.unifiedAddressing);
              printValueBool("Managed Memory:", deviceProp.managedMemory);
              printValueBool("Concurrent Managed Memory:", deviceProp.concurrentManagedAccess);
              printValueBool("Stream Priorities Supported:", deviceProp.streamPrioritiesSupported);
              printValueBool("Cooperative Launch:", deviceProp.cooperativeLaunch);
              printValueBool("Multi-Device Cooperative Launch:",
                             deviceProp.cooperativeMultiDeviceLaunch);

              // Kernel Execution Properties
              printHeader("Kernel Execution Properties:");
              printValueBool("Kernel Execution Timeout Enabled:",
                             deviceProp.kernelExecTimeoutEnabled);
#if CUDART_VERSION >= 11000
              printValue("Max Blocks Per MultiProcessor:",
                         std::to_string(deviceProp.maxBlocksPerMultiProcessor) + " blocks");
#endif

              // Hardware Features
              printHeader("Hardware Features:");
              printValueBool("ECC Enabled:", deviceProp.ECCEnabled);
              printValueBool("Is Multi-GPU Board:", deviceProp.isMultiGpuBoard);
              printValueBool("Can Map Host Memory:", deviceProp.canMapHostMemory);
              printValueBool("Can Use Host Pointer For Registered Memory:",
                             deviceProp.canUseHostPointerForRegisteredMem);
              if (deviceProp.major >= 7)
              {
                     printValueBool("Tensor Core Support:", true);
                     printValue("Tensor Core Generation:",
                                deviceProp.major == 7 ? "First Gen (Volta/Turing)" : deviceProp.major == 8 ? "Second Gen (Ampere)"
                                                                                 : deviceProp.major == 9   ? "Third Gen (Hopper)"
                                                                                                           : "Unknown");
              }
              else
              {
                     printValueBool("Tensor Core Support:", false);
              }

              // PCI Information
              printHeader("PCI Information:");
              printValue("PCI Bus ID:", std::to_string(deviceProp.pciBusID));
              printValue("PCI Device ID:", std::to_string(deviceProp.pciDeviceID));
              printValue("PCI Domain ID:", std::to_string(deviceProp.pciDomainID));

              // Persisting L2 Cache Properties
#if CUDART_VERSION >= 11000
              printHeader("Persisting L2 Cache Properties:");
              printValue("Persisting L2 Cache Max Size:",
                         formatMemorySize(deviceProp.persistingL2CacheMaxSize));
#endif

              // Performance Metrics
              printHeader("Performance Metrics:");

              // Calculate theoretical memory bandwidth
              printValue("Theoretical Memory Bandwidth:",
                         (std::stringstream() << std::fixed << std::setprecision(2)
                                              << (2.0f * (deviceProp.memoryClockRate * 1000.0f) *
                                                  (deviceProp.memoryBusWidth / 8.0f) / 1.0e9f) // Memory bandwidth = 2 * (memory clock rate) * (memory bus width / 8) / 1e9
                                              << " GB/s")
                             .str());

              // Calculate theoretical single-precision floating-point performance (TFLOPS)
              // Single-precision TFLOPS = 2 * (core clock rate) * (number of cormance (TFLOPS)
              printValue("Theoretical Single-Precision Performance:", formatTFlops((2.0f * deviceProp.clockRate * 1e-6f *
                                                                                    getCudaCoresPerSM(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount / 1000.0f)));

              // Calculate theoretical double-precision floating-point performance (TFLOPS)
              // Note: This is a rough estimate, actual ratio may vary by architecture
              // Double-precision TFLOPS = Single-precision TFLOPS / 2
              printValue("Theoretical Double-Precision Performance:", formatTFlops((2.0f * deviceProp.clockRate * 1e-6f *
                                                                                    getCudaCoresPerSM(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount / 1000.0f) /
                                                                                   2.0f));

              // Calculate SM utilization
              // SM utilization = (max threads per SM / max threads per block) * 100
              printValue("SM Utilization:", formatPercentage((static_cast<float>(deviceProp.maxThreadsPerMultiProcessor) / deviceProp.maxThreadsPerBlock) * 100.0f));

              printf("\n%s%s================================================%s\n",
                     BOLDMAGENTA, BOLD, RESET);
       }

       return EXIT_SUCCESS;
}